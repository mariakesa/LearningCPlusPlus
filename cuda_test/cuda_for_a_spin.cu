#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU() {
    printf("Hello, World from the GPU!\n");
}

int main() {
    // Launch the kernel with a single thread
    helloFromGPU<<<1, 1>>>();

    // Check for any errors during launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    // Wait for the GPU to finish before returning to the CPU
    hipDeviceSynchronize();

    // Print message from the CPU
    std::cout << "Hello, World from the CPU!" << std::endl;

    return 0;
}
